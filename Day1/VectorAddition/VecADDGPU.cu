#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
 

__global__
void vecAddkernel( float* A,  float* B, float* C, int n) {
	int i = threadIdx.x + blockDim.x + blockIdx.x;
	if (i < n) {
		C[i] = A[i] + B[i];
	}
}

int main() {
	const int n = 10;
	float A[n], B[n], C[n];
	float* A_d, * B_d, * C_d;

	int size = n * sizeof(float);
	
	hipMalloc(&A_d, size);
	hipMalloc(&B_d, size);
	hipMalloc(&C_d, size);

	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);


	vecAddkernel <<<ceil(n/256.0), 256 >> > (A_d, B_d, C_d, n);

	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	return 0;
}