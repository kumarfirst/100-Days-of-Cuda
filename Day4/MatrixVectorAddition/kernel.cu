#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <random>
#include <ctime>


__global__ void MatrixAddKernel(float* M, float* N, float* P, int Width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < Width) && (col < Width)) {
        float Pvalue = 0;
        Pvalue += M[row * Width + col] + N[row];
        P[row * Width + col] = Pvalue;
    }
}

void printMatrix(const char* name, float* matrix, int width) {
    printf("%s:\n", name);
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%.2f\t", matrix[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Fill a matrix with random integer values
void fillRandomMatrix(float* matrix, int numofelements) {
    for (int i = 0; i < numofelements; i++) {
        // Generate random values between 1 and 10
        matrix[i] = (rand() % 10 + 1);
    }
}

int main() {
    const int Width = 3; // assume its a 3*3 matrix
    int numofelements = Width * Width;
    int matrixsize = Width * Width * sizeof(float);
    int vectorsize = Width * sizeof(float);

    // Allocate memory for matrices on CPU
    float* h_M = (float*)malloc(matrixsize);   // First input matrix
    float* h_N = (float*)malloc(vectorsize);   // Second input matrix
    float* h_P = (float*)malloc(matrixsize);   // Result matrix

    // Initialize random seed
    srand(time(NULL));

    // Fill matrices with random values
    fillRandomMatrix(h_M, numofelements);
    fillRandomMatrix(h_N, Width);

    // Allocate memory for matrices on GPU
    float* d_M, * d_N, * d_P;
    hipMalloc(&d_M, matrixsize);
    hipMalloc(&d_N, vectorsize);
    hipMalloc(&d_P, matrixsize);

    // Copy input matrices from CPU to GPU
    hipMemcpy(d_M, h_M, matrixsize, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, vectorsize, hipMemcpyHostToDevice);

    // Configure thread layout (9 threads in a 3x3 block)
    dim3 blockSize(3, 3);  // 3x3 = 9 threads per block
    dim3 gridSize(1, 1);   // Only need 1 block for a 3x3 matrix

    // Run the kernel on the GPU
    MatrixAddKernel << <gridSize, blockSize >> > (d_M, d_N, d_P, Width);

    // Wait for GPU to finish
    hipDeviceSynchronize();


    // Copy the result back from GPU to CPU
    hipMemcpy(h_P, d_P, matrixsize, hipMemcpyDeviceToHost);

    // Display the matrices
    printMatrix("Matrix M", h_M, Width);
    printMatrix("Matrix N", h_N, Width);
    printMatrix("Result P = M × N", h_P, Width);

    // Clean up
    free(h_M);
    free(h_N);
    free(h_P);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}